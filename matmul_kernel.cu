#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <tuple>
#include <float_grad.h>
#include <hip/hip_vector_types.h>

// CUDA kernel
template <typename FloatType=float>
__global__ void matmul_kernel(
        const float* A, 
        const float* B, 
        float* C, 
        int M, int N, int K) {

    const FloatType* A_ptr = reinterpret_cast<const FloatType*>(A);
    const FloatType* B_ptr = reinterpret_cast<const FloatType*>(B);
    FloatType* C_ptr = reinterpret_cast<FloatType*>(C);
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        FloatType sum(0.0f);
        for (int k = 0; k < K; ++k) {
            sum += A_ptr[row * K + k] * B_ptr[k * N + col];
        }
        C_ptr[row * N + col] = sum;
    }
}

// Launcher function (visible to PyTorch)
torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.dtype() == torch::kFloat32, "A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "B must be float32");
    TORCH_CHECK(A.device().is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.device().is_cuda(), "B must be a CUDA tensor");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);
    TORCH_CHECK(B.size(0) == K, "A and B dimensions mismatch");

    auto C = torch::zeros({M, N}, A.options());

    dim3 blockDim(16, 16);
    dim3 gridDim((N + 15) / 16, (M + 15) / 16);

    matmul_kernel<<<gridDim, blockDim>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, N, K
    );

    return C;
}

torch::Tensor matmul_cuda_floatgrad(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.dtype() == torch::kFloat32, "A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "B must be float32");
    TORCH_CHECK(A.device().is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.device().is_cuda(), "B must be a CUDA tensor");
    
    // Check A has 3 dimensions and the last dimension is 2
    // Check B has 3 dimensions and the last dimension is 2
    TORCH_CHECK(A.dim() == 3 && A.size(2) == 2, "A must be a 3D tensor with last dimension 2");
    TORCH_CHECK(B.dim() == 3 && B.size(2) == 2, "B must be a 3D tensor with last dimension 2");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);
    TORCH_CHECK(B.size(0) == K, "A and B dimensions mismatch");

    auto C = torch::zeros({M, N, 2}, A.options());

    dim3 blockDim(16, 16);
    dim3 gridDim((N + 15) / 16, (M + 15) / 16);

    matmul_kernel<FloatGrad><<<gridDim, blockDim>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, N, K
    );

    return C;
}

// If needed: return a tuple
std::tuple<torch::Tensor> matmul_cuda_tuple(torch::Tensor A, torch::Tensor B) {
    auto C = matmul_cuda(A, B);
    return std::make_tuple(C);
}

