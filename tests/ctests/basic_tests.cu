#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "test_utils.h"

TEST(BasicTests, ScalarConstructor) {
    float a_data = 3.0f, a_grad = 1.0f;
    FloatGrad<float> a(a_data, a_grad);

    float b_data[2] = {4.0f, 5.0f};
    float b_grad[2] = {2.0f, 3.0f};

    FloatGrad<float> b0(b_data[0], b_grad[0]);
    FloatGrad<float> b1(b_data[1], b_grad[1]);
    FloatGrad<float2> b = make_float2(b0, b1);

    EXPECT_TRUE(float_eq(b.x, FloatGrad<float>(4.0f, 2.0f)));
    EXPECT_TRUE(float_eq(b.y, FloatGrad<float>(5.0f, 3.0f)));

}

TEST(BasicTests, ScalarOperators) {
    float a_data = 3.0f, a_grad = 1.0f;
    FloatGrad<float> a(a_data, a_grad);

    const float b_data = 4.0f;
    const float b_grad = 2.0f;
    FloatGradRef<const float> b(&b_data, &b_grad);

    float c_data = 5.0f, c_grad = 3.0f;
    FloatGradRef<float> c(&c_data, &c_grad);

    c = a + b;

    EXPECT_TRUE(float_eq(c, FloatGrad<float>(7.0f, 3.0f)));

    FloatGradRef<float> d(&a_data, &a_grad);
    d += b;

    EXPECT_TRUE(float_eq(d, c));

}

TEST(BasicTests, VectorConstructor) {
    const float2 a_data = make_float2(3.0f, 4.0f);
    const float2 a_grad = make_float2(1.0f, 2.0f);

    FloatGradRef<const float2> a(&a_data, &a_grad);

    float b_data[2] = {4.0f, 5.0f};
    float b_grad[2] = {2.0f, 3.0f};

    FloatGrad<float> b0(b_data[0], b_grad[0]);
    FloatGrad<float> b1(b_data[1], b_grad[1]);
    FloatGrad<float2> b = make_float2(b0, b1);

    EXPECT_TRUE(float_eq(b.x, FloatGrad<float>(4.0f, 2.0f)));
    EXPECT_TRUE(float_eq(b.y, FloatGrad<float>(5.0f, 3.0f)));

    float4 c_data = make_float4(6.0f, 7.0f, 8.0f, 9.0f);
    float4 c_grad = make_float4(4.0f, 5.0f, 6.0f, 7.0f);
    FloatGrad<float4> c(c_data, c_grad);
    FloatGrad<float4> d(c);

}

TEST(BasicTests, VectorRefConstructor) {
    float a_data = 3.0f, a_grad = 1.0f;
    FloatGrad<float> a(0.0f, 0.0f);

    a = a_data;

    EXPECT_TRUE(float_eq(a, FloatGrad<float>(3.0f, 0.0f)));


}

TEST(BasicTests, ArrayCompoundOperator) {
    float a_data[2] = {1.0f, 2.0f};
    float a_grad[2] = {0.1f, 0.2f};

    float b_data = 5.0f;
    float b_grad = 0.5f;
    FloatGradArray<float> a(a_data, a_grad);
    FloatGradRef<float> b(a_data, a_grad);
    FloatGradRef<float> c = a[0];

    c += b;
    a[0] += b;
}

