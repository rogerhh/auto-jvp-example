#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "hip/hip_vector_types.h"
#include "test_utils.h"

TEST(FloatGradFloat2, MakeFloat2) {
    FloatGrad<float> ax(1.0f, 0.1f);
    FloatGrad<float> ay(2.0f, 0.2f);
    FloatGrad<float2> a = make_float2(ax, ay);

    EXPECT_TRUE(float_eq(a, FloatGrad<float2>(make_float2(1.0f, 2.0f), make_float2(0.1f, 0.2f))));

    FloatGrad<float> bx(3.0f, 0.3f);
    FloatGrad<float2> b = make_float2(bx, 4.0f);

    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(make_float2(3.0f, 4.0f), make_float2(0.3f, 0.0f))));

}

TEST(FloatGradFloat2, VectorElementAccess) {
    float2 a_data = make_float2(1.0f, 2.0f);
    float2 a_grad = make_float2(0.1f, 0.2f);
    float2 b_data = make_float2(3.0f, 4.0f);
    float2 b_grad = make_float2(0.3f, 0.4f);

    FloatGrad<float2> a(a_data, a_grad);
    FloatGrad<float2> b(b_data, b_grad);

    EXPECT_TRUE(float_eq(a.x, FloatGrad<float>(1.0f, 0.1f)));
    EXPECT_TRUE(float_eq(a.y, FloatGrad<float>(2.0f, 0.2f)));

    a.x += b.x;

    EXPECT_TRUE(float_eq(a, FloatGrad<float2>(float2{4.0f, 2.0f}, float2{0.4f, 0.2f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(float2{3.0f, 4.0f}, float2{0.3f, 0.4f})));

    a.y *= b.y;

    EXPECT_TRUE(float_eq(a, FloatGrad<float2>(float2{4.0f, 8.0f}, float2{0.4f, 1.6f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(float2{3.0f, 4.0f}, float2{0.3f, 0.4f})));

    FloatGrad<const float2> c(a_data, a_grad);
    FloatGradRef<const float2> d(&b_data, &b_grad);

    EXPECT_TRUE(float_eq(c.x, FloatGrad<float>(1.0f, 0.1f)));
    EXPECT_TRUE(float_eq(c.y, FloatGrad<float>(2.0f, 0.2f)));
    EXPECT_TRUE(float_eq(d.x, FloatGrad<float>(3.0f, 0.3f)));
    EXPECT_TRUE(float_eq(d.y, FloatGrad<float>(4.0f, 0.4f)));
}

TEST(FloatGradFloat2, VectorOperators) {
    float2 a_data = make_float2(1.0f, 2.0f);
    float2 a_grad = make_float2(0.1f, 0.2f);
    float2 b_data = make_float2(3.0f, 4.0f);
    float2 b_grad = make_float2(0.3f, 0.4f);

    FloatGrad<float2> a(a_data, a_grad);
    FloatGrad<float2> b(b_data, b_grad);

    auto c = a + b;

    EXPECT_TRUE(float_eq(c, FloatGrad<float2>(make_float2(4.0f, 6.0f), 
                                              make_float2(0.4f, 0.6f))));

}

TEST(FloatGradFloat2, VectorArray) {
    float2 a_data[10];
    float2 a_grad[10];
    float2 b_data[10];
    float2 b_grad[10];

    FloatGradArray<float2> a(a_data, a_grad);
    FloatGradArray<float2> b(b_data, b_grad);

    for (int i = 0; i < 10; ++i) {
        a_data[i] = make_float2(float(i + 1), float(i + 2));
        a_grad[i] = make_float2(float(0.1f * (i + 1)), float(0.2f * (i + 1)));
        b_data[i] = make_float2(float(i + 3), float(i + 4));
        b_grad[i] = make_float2(float(0.3f * (i + 1)), float(0.4f * (i + 1)));
    }

    const float2* c_data = a_data;
    const float2* c_grad = a_grad;
    const float2* d_data = b_data;
    const float2* d_grad = b_grad;

    FloatGradArray<const float2> c(c_data, c_grad);
    FloatGradArray<const float2> d(d_data, d_grad);

    int i = 5;
    FloatGrad<float2> e = c[i] + d[i];
    EXPECT_TRUE(float_eq(e, FloatGrad<float2>(make_float2(14.0f, 16.0f), 
                                              make_float2(2.4f, 3.6f))));

}
