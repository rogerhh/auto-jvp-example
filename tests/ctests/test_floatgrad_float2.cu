#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "hip/hip_vector_types.h"
#include "test_utils.h"

TEST(FloatGradFloat2, MakeFloat2) {
    FloatGrad<float> ax(1.0f, 0.1f);
    FloatGrad<float> ay(2.0f, 0.2f);
    FloatGrad<float2> a = make_float2(ax, ay);

    EXPECT_TRUE(float_eq(a, FloatGrad<float2>(make_float2(1.0f, 2.0f), make_float2(0.1f, 0.2f))));

    FloatGrad<float> bx(3.0f, 0.3f);
    FloatGrad<float2> b = make_float2(bx, 4.0f);

    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(make_float2(3.0f, 4.0f), make_float2(0.3f, 0.0f))));

}

TEST(FloatGradFloat2, VectorElementAccess) {
    float2 a_data = make_float2(1.0f, 2.0f);
    float2 a_grad = make_float2(0.1f, 0.2f);
    float2 b_data = make_float2(3.0f, 4.0f);
    float2 b_grad = make_float2(0.3f, 0.4f);

    FloatGrad<float2> a(a_data, a_grad);
    FloatGrad<float2> b(b_data, b_grad);

    EXPECT_TRUE(float_eq(a.x, FloatGrad<float>(1.0f, 0.1f)));
    EXPECT_TRUE(float_eq(a.y, FloatGrad<float>(2.0f, 0.2f)));

    a.x += b.x;

    EXPECT_TRUE(float_eq(a, FloatGrad<float2>(float2{4.0f, 2.0f}, float2{0.4f, 0.2f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(float2{3.0f, 4.0f}, float2{0.3f, 0.4f})));

    a.y *= b.y;

    EXPECT_TRUE(float_eq(a, FloatGrad<float2>(float2{4.0f, 8.0f}, float2{0.4f, 1.6f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(float2{3.0f, 4.0f}, float2{0.3f, 0.4f})));
}

TEST(FloatGradFloat2, VectorOperators) {
    float2 a_data = make_float2(1.0f, 2.0f);
    float2 a_grad = make_float2(0.1f, 0.2f);
    float2 b_data = make_float2(3.0f, 4.0f);
    float2 b_grad = make_float2(0.3f, 0.4f);

    FloatGrad<float2> a(a_data, a_grad);
    FloatGrad<float2> b(b_data, b_grad);

    auto c = a + b;

    EXPECT_TRUE(float_eq(c, FloatGrad<float2>(make_float2(4.0f, 6.0f), 
                                              make_float2(0.4f, 0.6f))));

}
