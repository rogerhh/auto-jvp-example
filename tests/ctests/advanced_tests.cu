#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "test_utils.h"
#include "hip/hip_vector_types.h"

template <typename T>
void print_type() {
    static_assert(always_false<T>::value, "This is a placeholder to ensure the function is not optimized out.");
}

FloatGrad<float4> transformPoint4x4(const FloatGradRef<const float3>& p, const FloatGradArray<const float> matrix)
{
    FloatGrad<float4> transformed = make_float4(
        matrix[0] * p.x + matrix[4] * p.y + matrix[8] * p.z + matrix[12],
        matrix[1] * p.x + matrix[5] * p.y + matrix[9] * p.z + matrix[13], 
        matrix[2] * p.x + matrix[6] * p.y + matrix[10] * p.z + matrix[14],
        matrix[3] * p.x + matrix[7] * p.y + matrix[11] * p.z + matrix[15]
    );
    return transformed;
}


TEST(AdvancedTest, TransformPoint4x4) {
    float3 p_data = make_float3(1.0f, 2.0f, 3.0f);
    float3 p_grad = make_float3(0.1f, 0.2f, 0.3f);
    FloatGradRef<const float3> p(&p_data, &p_grad);

    float matrix_data[16];
    float matrix_grad[16];

    for (int i = 0; i < 16; ++i) {
        matrix_data[i] = static_cast<float>(i + 1);
        matrix_grad[i] = static_cast<float>((i + 1) * 0.2);
    }
    FloatGradArray<const float> matrix(matrix_data, matrix_grad);

    float4 transformed_data = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float4 transformed_grad = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    FloatGradRef<float4> transformed(&transformed_data, &transformed_grad);
    FloatGrad<float4> test(transformed_data, transformed_grad);

    transformed = transformPoint4x4(p, matrix);

}
