#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "test_utils.h"

TEST(FloatGradTest, ScalarArrayOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);

    for(int i = 0; i < 10; i++) {
        FloatGrad<float> ai{a_data[i], a_grad[i]};
        FloatGrad<float> bi{b_data[i], b_grad[i]};

        EXPECT_TRUE(float_eq(ai, a[i]));
        EXPECT_TRUE(float_eq(bi, b[i]));
        EXPECT_TRUE(float_eq(ai + bi, a[i] + b[i]));
        EXPECT_TRUE(float_eq(ai - bi, a[i] - b[i]));
        EXPECT_TRUE(float_eq(ai * bi, a[i] * b[i]));
        EXPECT_TRUE(float_eq(ai / bi, a[i] / b[i]));
        EXPECT_TRUE(float_eq(sqrtf(bi), sqrtf(b[i])));

        EXPECT_TRUE(!((ai < bi) ^ (a[i] < b[i])));
        EXPECT_TRUE(!((ai > bi) ^ (a[i] > b[i])));
        EXPECT_TRUE(!((ai <= bi) ^ (a[i] <= b[i])));
        EXPECT_TRUE(!((ai >= bi) ^ (a[i] >= b[i])));
    }

}

TEST(FloatGradTest, ScalarArrayCompoundOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};
    float c_data[10] = {-5.0f, -4.0f, -3.0f, -2.0f, -1.0f,
                            0.0f, 1.0f, 2.0f, 3.0f, 4.0f};
    float c_grad[10] = {-1.0f, -0.9f, -0.8f, -0.7f, -0.6f,
                            -0.5f, -0.4f, -0.3f, -0.2f, -0.1f};

    float c_copy_data[10];
    float c_copy_grad[10];

    for(int i = 0; i < 10; i++) {
        c_copy_data[i] = c_data[i];
        c_copy_grad[i] = c_grad[i];
    }

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);
    FloatGradArray<float> c(c_data, c_grad);

    for(int i = 0; i < 10; i++) {
        c[i] += a[i] * b[i];

        EXPECT_TRUE(float_eq(c[i], 
                    FloatGrad<float>(c_copy_data[i] + a_data[i] * b_data[i],
                    c_copy_grad[i] + a_grad[i] * b_data[i] + a_data[i] * b_grad[i])));
    }

}

TEST(FloatGradTest, ConstScalarArrayOperators) {
    const float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    const float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    const float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    const float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};

    FloatGradArray<const float> a(a_data, a_grad);
    FloatGradArray<const float> b(b_data, b_grad);

    for(int i = 0; i < 10; i++) {
        FloatGrad<float> ai{a_data[i], a_grad[i]};
        FloatGrad<float> bi{b_data[i], b_grad[i]};

        EXPECT_TRUE(float_eq(ai, a[i]));
        EXPECT_TRUE(float_eq(bi, b[i]));
        EXPECT_TRUE(float_eq(ai + bi, a[i] + b[i]));
        EXPECT_TRUE(float_eq(ai - bi, a[i] - b[i]));
        EXPECT_TRUE(float_eq(ai * bi, a[i] * b[i]));
        EXPECT_TRUE(float_eq(ai / bi, a[i] / b[i]));
        EXPECT_TRUE(float_eq(sqrtf(bi), sqrtf(b[i])));

        EXPECT_TRUE(!((ai < bi) ^ (a[i] < b[i])));
        EXPECT_TRUE(!((ai > bi) ^ (a[i] > b[i])));
        EXPECT_TRUE(!((ai <= bi) ^ (a[i] <= b[i])));
        EXPECT_TRUE(!((ai >= bi) ^ (a[i] >= b[i])));
    }

}

TEST(FloatGradTest, ConstScalarArrayCompoundOperators) {
    const float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    const float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    const float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    const float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};
    float c_data[10] = {-5.0f, -4.0f, -3.0f, -2.0f, -1.0f,
                            0.0f, 1.0f, 2.0f, 3.0f, 4.0f};
    float c_grad[10] = {-1.0f, -0.9f, -0.8f, -0.7f, -0.6f,
                            -0.5f, -0.4f, -0.3f, -0.2f, -0.1f};

    float c_copy_data[10];
    float c_copy_grad[10];

    for(int i = 0; i < 10; i++) {
        c_copy_data[i] = c_data[i];
        c_copy_grad[i] = c_grad[i];
    }

    FloatGradArray<const float> a(a_data, a_grad);
    FloatGradArray<const float> b(b_data, b_grad);
    FloatGradArray<float> c(c_data, c_grad);

    for(int i = 0; i < 10; i++) {
        c[i] -= a[i] * b[i];

        EXPECT_TRUE(float_eq(c[i], 
                    FloatGrad<float>(c_copy_data[i] - a_data[i] * b_data[i],
                    c_copy_grad[i] - (a_grad[i] * b_data[i] + a_data[i] * b_grad[i]))));
    }

}

TEST(FloatGradTest, ArrayElementAssignment) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                        6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                        0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                        12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                        0.05f, 0.04f, 0.03f, 0.02f, 0.01f};


    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);

    for(int i = 0; i < 10; i++) {
        // volatile auto tmp = a[i] = b[i];
        FloatGradRef<float> aref(&a_data[i], &a_grad[i]);
        FloatGradRef<float> bref(&b_data[i], &b_grad[i]);
        aref = bref;

    }

    for(int i = 0; i < 10; i++) {
        EXPECT_TRUE(float_eq(a[i], b[i]));
    }


    float c_data = -5.0f;
    float c_grad = -1.0f;

    FloatGradRef<float> c(&a_data[0], &a_grad[0]);

    float d_data = 3.0f;
    float d_grad = 0.5f;

    FloatGradRef<float> d(&d_data, &d_grad);

    c = d;

}

TEST(FloatGradTest, ArrayElementAssignmentVec2) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                        6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                        0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                        12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                        0.05f, 0.04f, 0.03f, 0.02f, 0.01f};


    FloatGradArray<float2> a((float2*) a_data, (float2*) a_grad);
    FloatGradArray<float2> b((float2*) b_data, (float2*) b_grad);

    for(int i = 0; i < 5; i++) {
        a[i] = b[i];
    }

    for(int i = 0; i < 5; i++) {
        EXPECT_TRUE(float_eq(a[i], b[i]));
    }


}

TEST(FloatGradTest, ArrayOffset) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                        6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                        0.5f, 0.4f, 0.3f, 0.2f, 0.1f};

    FloatGradArray<float> a(a_data, a_grad);

    int offset = 5;

    FloatGradArray<float> b = a + 5;

    EXPECT_TRUE(float_eq(b[0], a[5]));

}

