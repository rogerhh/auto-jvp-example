#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "hip/hip_vector_types.h"

TEST(FloatGradHelperMathTest, VectorConstruct) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float2> a2 = make_float2(a);
}

