#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "test_utils.h"
#include "hip/hip_vector_types.h"

TEST(FloatGradHelperMathTest, FloatOperations) {
    float test = fminf(0.3, 0.4);
    EXPECT_FLOAT_EQ(test, 0.3f) << "Expected: 0.3, Got: " << test;

    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 1.0f);

    FloatGrad<float> c = fminf(a, b);
    EXPECT_TRUE(float_eq(c, a));

    c = fminf(FloatGrad<float>(1.0f, 10.0f), a);
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(1.0f, 10.0f)));

    c = fmaxf(-b, a);
    EXPECT_TRUE(float_eq(c, a));

    c = sqrtf(b);
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(2.0f, 0.25f)));

    c = rsqrtf(b);
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(0.5f, -1.0f / 16.0f)));

    FloatGrad<float> d(3.4f, 0.25f);
    EXPECT_TRUE(float_eq(floorf(d), FloatGrad<float>(3.0f, 0.0f)));
    EXPECT_TRUE(float_eq(ceilf(d), FloatGrad<float>(4.0f, 0.0f)));
    EXPECT_TRUE(float_eq(roundf(d), FloatGrad<float>(3.0f, 0.0f)));
    EXPECT_TRUE(float_eq(truncf(d), FloatGrad<float>(3.0f, 0.0f)));
    EXPECT_TRUE(float_eq(fmodf(d, 2.0f), FloatGrad<float>(1.4f, 0.25f)));
    EXPECT_TRUE(float_eq(fmodf(d, 1.1f), FloatGrad<float>(0.1f, 0.25f)));
    EXPECT_TRUE(float_eq(fabs(d), FloatGrad<float>(3.4f, 0.25f)));

    FloatGrad<float> e(-5.6f, 1.0f);
    EXPECT_TRUE(float_eq(floorf(e), FloatGrad<float>(-6.0f, 0.0f)));
    EXPECT_TRUE(float_eq(ceilf(e), FloatGrad<float>(-5.0f, 0.0f)));
    EXPECT_TRUE(float_eq(roundf(e), FloatGrad<float>(-6.0f, 0.0f)));
    EXPECT_TRUE(float_eq(truncf(e), FloatGrad<float>(-5.0f, 0.0f)));
    EXPECT_TRUE(float_eq(fmodf(e, 2.0f), FloatGrad<float>(-1.6f, 1.0f)));
    EXPECT_TRUE(float_eq(fabs(e), FloatGrad<float>(5.6f, -1.0f)));

    FloatGrad<float> f(2.0f, 1.0f);
    EXPECT_TRUE(float_eq(fmodf(4.4f, f), FloatGrad<float>(0.4f, -2.0f)));

}

TEST(FloatGradHelperMathTest, MakeFloat2) {
    float3 a_data = make_float3(1.0f, 2.0f, 3.0f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);

    FloatGrad<float3> a(a_data, a_grad);

    FloatGrad<float2> b = make_float2(a.z);

    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(make_float2(3.0f, 3.0f), 
                                              make_float2(0.3f, 0.3f))));

    FloatGrad<float2> c = make_float2(a);
    EXPECT_TRUE(float_eq(c, FloatGrad<float2>(make_float2(1.0f, 2.0f), 
                                              make_float2(0.1f, 0.2f))));
}

TEST(FloatGradHelperMathTest, MakeFloat3) {
    FloatGrad<float> a(1.0f, 0.1f);
    FloatGrad<float3> b = 3.0f * make_float3(a);
    EXPECT_TRUE(float_eq(b, FloatGrad<float3>(make_float3(3.0f, 3.0f, 3.0f), 
                                              make_float3(0.3f, 0.3f, 0.3f))));

    float2 c = make_float2(2.0f, 4.0f);
    FloatGrad<float3> d = make_float3(c, a);
    EXPECT_TRUE(float_eq(d, FloatGrad<float3>(make_float3(2.0f, 4.0f, 1.0f), 
                                              make_float3(0.0f, 0.0f, 0.1f))));

}

TEST(FloatGradHelperMathTest, MakeFloat4) {
    FloatGrad<float> a(1.0f, 0.1f);
    FloatGrad<float4> b = 3.0f * make_float4(a);
    EXPECT_TRUE(float_eq(b, FloatGrad<float4>(make_float4(3.0f, 3.0f, 3.0f, 3.0f), 
                                              make_float4(0.3f, 0.3f, 0.3f, 0.3f))));

    float3 c = make_float3(2.0f, 4.0f, 0.1f);
    FloatGrad<float4> d = make_float4(c, a);
    EXPECT_TRUE(float_eq(d, FloatGrad<float4>(make_float4(2.0f, 4.0f, 0.1f, 1.0f), 
                                              make_float4(0.0f, 0.0f, 0.0f, 0.1f))));

}

TEST(FloatGradHelperMathTest, ArithmeticOperators) {
    float4 a_data = make_float4(1.0f, 2.0f, 3.0f, 4.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.3f, 0.4f);

    float4 b_data = make_float4(-2.0f, -4.0f, -6.0f, -8.0f);
    float4 b_grad = make_float4(-0.2f, -0.4f, -0.6f, -0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    EXPECT_TRUE(float_eq(-b, FloatGrad<float4>(make_float4(2.0f, 4.0f, 6.0f, 8.0f),
                                               make_float4(0.2f, 0.4f, 0.6f, 0.8f))));

    FloatGrad<float4> c = a + b;

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(-1.0f, -2.0f, -3.0f, -4.0f), 
                                              make_float4(-0.1f, -0.2f, -0.3f, -0.4f))));

    c = a - b_grad;

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(1.2f, 2.4f, 3.6f, 4.8f), 
                                              make_float4(0.1f, 0.2f, 0.3f, 0.4f))));

    FloatGrad<float3> d = make_float3(c);
    FloatGrad<float3> e = make_float3(a.w);

    d += e;

    EXPECT_TRUE(float_eq(d, FloatGrad<float3>(make_float3(5.2f, 6.4f, 7.6f), 
                                              make_float3(0.5f, 0.6f, 0.7f))));

    float2 f_data = make_float2(1.0f, 2.0f);
    float2 f_grad = make_float2(0.1f, 0.3f);

    FloatGrad<float2> f(f_data, f_grad);

    float2 g_data = make_float2(3.0f, 4.0f);
    float2 g_grad = make_float2(0.2f, 0.4f);

    FloatGrad<float2> g(g_data, g_grad);

    EXPECT_TRUE(float_eq(f * g_grad, FloatGrad<float2>(make_float2(0.2f, 0.8f), 
                                                       make_float2(0.02f, 0.12f))));

    EXPECT_TRUE(float_eq(f * g, FloatGrad<float2>(make_float2(3.0f, 8.0f), 
                                                  make_float2(0.5f, 2.0f))));

    EXPECT_TRUE(float_eq(g / f, FloatGrad<float2>(make_float2(3.0f, 2.0f), 
                                                  make_float2(-0.1f / 1, -0.4f / 4))));

}

TEST(FloatGradHelperMathTest, MinMaxFunctions) {
    float4 a_data = make_float4(1.0f, 2.0f, 3.0f, 4.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.3f, 0.4f);

    float4 b_data = make_float4(-2.0f, 3.0f, 2.5f, 5.0f);
    float4 b_grad = make_float4(-0.2f, -0.4f, -0.6f, -0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    FloatGrad<float4> c = fminf(a, b);

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(-2.0f, 2.0f, 2.5f, 4.0f), 
                                              make_float4(-0.2f, 0.2f, -0.6f, 0.4f))));

    float4 d = make_float4(1.0f, 1.0f, -3.0f, 5.0f);

    FloatGrad<float4> e = fminf(c, d);

    EXPECT_TRUE(float_eq(e, FloatGrad<float4>(make_float4(-2.0f, 1.0f, -3.0f, 4.0f), 
                                              make_float4(-0.2f, 0.0f, -0.0f, 0.4f))));

    FloatGrad<float4> f = fmaxf(a, b);

    EXPECT_TRUE(float_eq(f, FloatGrad<float4>(make_float4(1.0f, 3.0f, 3.0f, 5.0f), 
                                              make_float4(0.1f, -0.4f, 0.3f, -0.8f))));

    FloatGrad<float4> g = fmaxf(d, c);

    EXPECT_TRUE(float_eq(g, FloatGrad<float4>(make_float4(1.0f, 2.0f, 2.5f, 5.0f), 
                                              make_float4(0.0f, 0.2f, -0.6f, 0.0f))));

}

TEST(FloatGradHelperMathTest, LerpFunctions) {
    float4 a_data = make_float4(1.0f, 2.0f, 3.0f, 4.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.3f, 0.4f);

    float4 b_data = make_float4(2.0f, 4.0f, 6.0f, 8.0f);
    float4 b_grad = make_float4(0.2f, 0.4f, 0.6f, 0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    EXPECT_TRUE(float_eq(lerp(a.x, 3.0f, 0.5f), 
                         FloatGrad<float>(2.0f, 0.05f)));

    FloatGrad<float> t(0.1f, 1.0f);

    FloatGrad<float4> c = lerp(a, b, t);

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(1.1f, 2.2f, 3.3f, 4.4f), 
                                              make_float4(1.11f, 2.22f, 3.33f, 4.44f))));
}

TEST(FloatGradHelperMathTest, ClampFunctions) {
    float4 a_data = make_float4(11.0f, 2.5f, -1.0f, 4.0f);
    float4 a_grad = make_float4(-0.1f, -0.2f, 0.3f, -0.4f);

    float4 b_data = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float4 b_grad = make_float4(0.2f, 0.4f, 0.6f, 0.8f);

    float4 c_data = make_float4(10.0f, 10.0f, 10.0f, 10.0f);
    float4 c_grad = make_float4(0.1f, 0.2f, 0.3f, 0.4f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);
    FloatGrad<float4> c(c_data, c_grad);
    FloatGrad<float4> d = clamp(a, b, c);

    EXPECT_TRUE(float_eq(d, FloatGrad<float4>(make_float4(10.0f, 2.5f, 0.0f, 4.0f), 
                                              make_float4(0.1f, -0.2f, 0.6f, -0.4f))));

}

TEST(FloatGradHelperMathTest, DotFunctions) {
    float4 a_data = make_float4(11.0f, 2.5f, -1.0f, 4.0f);
    float4 a_grad = make_float4(-0.1f, -0.2f, 0.3f, -0.4f);

    float4 b_data = make_float4(1.0f, 2.0f, 3.0f, 4.0f);
    float4 b_grad = make_float4(0.2f, 0.4f, 0.6f, 0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    auto c = dot(a, b);

    EXPECT_TRUE(float_eq(c, FloatGrad<float>(29.0f, 4.6f)));

}

TEST(FloatGradHelperMathTest, LengthFunctions) {
    float3 a_data = make_float3(1.0f, 2.0f, 3.0f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);

    FloatGrad<float3> a(a_data, a_grad);

    auto c = length(a);

    EXPECT_TRUE(float_eq(c, FloatGrad<float>(sqrtf(14.0f), 0.5f * 2.8f / sqrtf(14.0f))));

    auto d = normalize(a);

    float a_norm_sq = dot(a_data, a_data);
    float rsqrt_a_norm_sq = rsqrtf(a_norm_sq);
    float3 d_grad_ref = rsqrt_a_norm_sq * a_grad 
                        - rsqrt_a_norm_sq / a_norm_sq * dot(a_data, a_grad) * a_data;

    EXPECT_TRUE(float_eq(d, FloatGrad<float3>(normalize(a_data), d_grad_ref)));

}

TEST(FloatGradHelperMathTest, FloorFunctions) {
    float3 a_data = make_float3(1.1f, 2.9f, 3.5f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);

    FloatGrad<float3> a(a_data, a_grad);

    FloatGrad<float3> b = floorf(a);

    EXPECT_TRUE(float_eq(b, FloatGrad<float3>(make_float3(1.0f, 2.0f, 3.0f), 
                                              make_float3(0.0f, 0.0f, 0.0f))));

}

TEST(FloatGradHelperMathTest, FracFunctions) {
    float3 a_data = make_float3(1.1f, 2.9f, 3.5f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);

    FloatGrad<float3> a(a_data, a_grad);

    FloatGrad<float3> b = fracf(a);

    EXPECT_TRUE(float_eq(b, FloatGrad<float3>(make_float3(0.1f, 0.9f, 0.5f), 
                                              make_float3(0.1f, 0.2f, 0.3f))));

}

TEST(FloatGradHelperMathTest, FmodFunctions) {
    float3 a_data = make_float3(1.1f, 2.9f, 3.5f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);
    float3 b_data = make_float3(0.3f, 0.2f, 0.7f);
    float3 b_grad = make_float3(0.2f, 0.4f, 0.5f);

    FloatGrad<float3> a(a_data, a_grad);
    FloatGrad<float3> b(b_data, b_grad);

    FloatGrad<float3> c = fmodf(a, b);

    EXPECT_TRUE(float_eq(c, FloatGrad<float3>(make_float3(0.2f, 0.1f, 0.0f), 
                                              make_float3(-0.5f, -5.4f, -2.2f))));

}

TEST(FloatGradHelperMathTest, FabsFunctions) {
    float3 a_data = make_float3(1.1f, -2.9f, 3.5f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);

    FloatGrad<float3> a(a_data, a_grad);
    FloatGrad<float3> b = fabs(a);

    EXPECT_TRUE(float_eq(b, FloatGrad<float3>(make_float3(1.1f, 2.9f, 3.5f), 
                                              make_float3(0.1f, -0.2f, 0.3f))));

}

TEST(FloatGradHelperMathTest, ReflectAutodiff) {
    float3 a_data = make_float3(1.1f, -2.9f, 3.5f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);
    float3 b_data = normalize(make_float3(0.3f, 0.2f, 0.7f));
    float3 b_grad = make_float3(0.2f, 0.4f, 0.5f);

    FloatGrad<float3> a(a_data, a_grad);
    FloatGrad<float3> b(b_data, b_grad);

    float3 c_data_ref = reflect(a_data, b_data);
    FloatGrad<float3> c = reflect(a, b);

    EXPECT_FLOAT_EQ(c_data_ref.x, c.x.data());
    EXPECT_FLOAT_EQ(c_data_ref.y, c.y.data());
    EXPECT_FLOAT_EQ(c_data_ref.z, c.z.data());

    float* x_ptr[6] = {&a_data.x, &a_data.y, &a_data.z, 
                       &b_data.x, &b_data.y, &b_data.z};
    float* g_ptr[6] = {&a_grad.x, &a_grad.y, &a_grad.z,
                       &b_grad.x, &b_grad.y, &b_grad.z};

    float3 c_grad_ref = make_float3(0.0f, 0.0f, 0.0f);

    float eps = 1e-4;   // Need to choose large enough epsilon to avoid numerical issues
    for(int i = 0; i < 6; i++) {
        float x_data_backup = *x_ptr[i];
        *x_ptr[i] += eps;
        float3 c_plus = reflect(a_data, b_data);
        *x_ptr[i] = x_data_backup - eps;
        float3 c_minus = reflect(a_data, b_data);
        *x_ptr[i] = x_data_backup;

        c_grad_ref += (c_plus - c_minus) / (2 * eps) * (*g_ptr[i]);
    }

    EXPECT_TRUE(float_eq(c.grad(), c_grad_ref, 1e-3)) << "Expected: " << c_grad_ref.x << ", " 
                                                      << c_grad_ref.y << ", " << c_grad_ref.z 
                                                      << " Got: " << c.grad().x << ", " 
                                                      << c.grad().y << ", " << c.grad().z;

}

TEST(FloatGradHelperMathTest, CrossAutodiff) {
    float3 a_data = make_float3(1.1f, -2.9f, 3.5f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);
    float3 b_data = normalize(make_float3(0.3f, 0.2f, 0.7f));
    float3 b_grad = make_float3(0.2f, 0.4f, 0.5f);

    FloatGrad<float3> a(a_data, a_grad);
    FloatGrad<float3> b(b_data, b_grad);

    float3 c_data_ref = cross(a_data, b_data);
    FloatGrad<float3> c = cross(a, b);

    EXPECT_FLOAT_EQ(c_data_ref.x, c.x.data());
    EXPECT_FLOAT_EQ(c_data_ref.y, c.y.data());
    EXPECT_FLOAT_EQ(c_data_ref.z, c.z.data());

    float* x_ptr[6] = {&a_data.x, &a_data.y, &a_data.z, 
                       &b_data.x, &b_data.y, &b_data.z};
    float* g_ptr[6] = {&a_grad.x, &a_grad.y, &a_grad.z,
                       &b_grad.x, &b_grad.y, &b_grad.z};

    float3 c_grad_ref = make_float3(0.0f, 0.0f, 0.0f);

    float eps = 1e-4;
    for(int i = 0; i < 6; i++) {
        float x_data_backup = *x_ptr[i];
        *x_ptr[i] += eps;
        float3 c_plus = cross(a_data, b_data);
        *x_ptr[i] = x_data_backup - eps;
        float3 c_minus = cross(a_data, b_data);
        *x_ptr[i] = x_data_backup;

        c_grad_ref += (c_plus - c_minus) / (2 * eps) * (*g_ptr[i]);
    }

    EXPECT_TRUE(float_eq(c.grad(), c_grad_ref, 1e-3)) << "Expected: " << c_grad_ref.x << ", " 
                                                      << c_grad_ref.y << ", " << c_grad_ref.z 
                                                      << " Got: " << c.grad().x << ", " 
                                                      << c.grad().y << ", " << c.grad().z;

}

TEST(FloatGradHelperMathTest, SmoothStepAutodiff) {
    float3 a_data = make_float3(1.1f, -2.9f, 3.5f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);
    float3 b_data = normalize(make_float3(0.3f, 0.2f, 0.7f));
    float3 b_grad = make_float3(0.2f, 0.4f, 0.5f);
    float3 d_data = make_float3(1.3f, -2.0f, 3.65f);
    float3 d_grad = make_float3(0.45f, 0.4f, -0.5f);

    FloatGrad<float3> a(a_data, a_grad);
    FloatGrad<float3> b(b_data, b_grad);
    FloatGrad<float3> d(d_data, d_grad);

    float3 c_data_ref = smoothstep(a_data, b_data, d_data);
    FloatGrad<float3> c = smoothstep(a, b, d);

    EXPECT_FLOAT_EQ(c_data_ref.x, c.x.data());
    EXPECT_FLOAT_EQ(c_data_ref.y, c.y.data());
    EXPECT_FLOAT_EQ(c_data_ref.z, c.z.data());

    float* x_ptr[9] = {&a_data.x, &a_data.y, &a_data.z, 
                       &b_data.x, &b_data.y, &b_data.z,
                       &d_data.x, &d_data.y, &d_data.z};
    float* g_ptr[9] = {&a_grad.x, &a_grad.y, &a_grad.z,
                       &b_grad.x, &b_grad.y, &b_grad.z,
                       &d_grad.x, &d_grad.y, &d_grad.z};

    float3 c_grad_ref = make_float3(0.0f, 0.0f, 0.0f);

    float eps = 1e-4;
    for(int i = 0; i < 9; i++) {
        float x_data_backup = *x_ptr[i];
        *x_ptr[i] += eps;
        float3 c_plus = smoothstep(a_data, b_data, d_data);
        *x_ptr[i] = x_data_backup - eps;
        float3 c_minus = smoothstep(a_data, b_data, d_data);
        *x_ptr[i] = x_data_backup;

        c_grad_ref += (c_plus - c_minus) / (2 * eps) * (*g_ptr[i]);
    }

    EXPECT_TRUE(float_eq(c.grad(), c_grad_ref, 1e-3)) << "Expected: " << c_grad_ref.x << ", " 
                                                      << c_grad_ref.y << ", " << c_grad_ref.z 
                                                      << " Got: " << c.grad().x << ", " 
                                                      << c.grad().y << ", " << c.grad().z;

}
