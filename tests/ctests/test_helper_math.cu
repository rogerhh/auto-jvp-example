#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "test_utils.h"
#include "hip/hip_vector_types.h"

TEST(FloatGradHelperMathTest, FloatOperations) {
    float test = fminf(0.3, 0.4);
    EXPECT_FLOAT_EQ(test, 0.3f) << "Expected: 0.3, Got: " << test;

    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 1.0f);

    FloatGrad<float> c = fminf(a, b);
    EXPECT_TRUE(float_eq(c, a));

    c = fminf(FloatGrad<float>(1.0f, 10.0f), a);
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(1.0f, 10.0f)));

    c = fmaxf(-b, a);
    EXPECT_TRUE(float_eq(c, a));

    c = sqrtf(b);
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(2.0f, 0.25f)));

    c = rsqrtf(b);
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(0.5f, -1.0f / 16.0f)));
}

TEST(FloatGradHelperMathTest, MakeFloat2) {
    float3 a_data = make_float3(1.0f, 2.0f, 3.0f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);

    FloatGrad<float3> a(a_data, a_grad);

    FloatGrad<float2> b = make_float2(a.z);

    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(make_float2(3.0f, 3.0f), 
                                              make_float2(0.3f, 0.3f))));

    FloatGrad<float2> c = make_float2(a);
    EXPECT_TRUE(float_eq(c, FloatGrad<float2>(make_float2(1.0f, 2.0f), 
                                              make_float2(0.1f, 0.2f))));
}

TEST(FloatGradHelperMathTest, MakeFloat3) {
    FloatGrad<float> a(1.0f, 0.1f);
    FloatGrad<float3> b = 3.0f * make_float3(a);
    EXPECT_TRUE(float_eq(b, FloatGrad<float3>(make_float3(3.0f, 3.0f, 3.0f), 
                                              make_float3(0.3f, 0.3f, 0.3f))));

    float2 c = make_float2(2.0f, 4.0f);
    FloatGrad<float3> d = make_float3(c, a);
    EXPECT_TRUE(float_eq(d, FloatGrad<float3>(make_float3(2.0f, 4.0f, 1.0f), 
                                              make_float3(0.0f, 0.0f, 0.1f))));

}

TEST(FloatGradHelperMathTest, MakeFloat4) {
    FloatGrad<float> a(1.0f, 0.1f);
    FloatGrad<float4> b = 3.0f * make_float4(a);
    EXPECT_TRUE(float_eq(b, FloatGrad<float4>(make_float4(3.0f, 3.0f, 3.0f, 3.0f), 
                                              make_float4(0.3f, 0.3f, 0.3f, 0.3f))));

    float3 c = make_float3(2.0f, 4.0f, 0.1f);
    FloatGrad<float4> d = make_float4(c, a);
    EXPECT_TRUE(float_eq(d, FloatGrad<float4>(make_float4(2.0f, 4.0f, 0.1f, 1.0f), 
                                              make_float4(0.0f, 0.0f, 0.0f, 0.1f))));

}

TEST(FloatGradHelperMathTest, ArithmeticOperators) {
    float4 a_data = make_float4(1.0f, 2.0f, 3.0f, 4.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.3f, 0.4f);

    float4 b_data = make_float4(-2.0f, -4.0f, -6.0f, -8.0f);
    float4 b_grad = make_float4(-0.2f, -0.4f, -0.6f, -0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    FloatGrad<float4> c = a + b;

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(-1.0f, -2.0f, -3.0f, -4.0f), 
                                              make_float4(-0.1f, -0.2f, -0.3f, -0.4f))));

    c = a - b_grad;

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(1.2f, 2.4f, 3.6f, 4.8f), 
                                              make_float4(0.1f, 0.2f, 0.3f, 0.4f))));

    FloatGrad<float3> d = make_float3(c);
    FloatGrad<float3> e = make_float3(a.w);

    d += e;

    EXPECT_TRUE(float_eq(d, FloatGrad<float3>(make_float3(5.2f, 6.4f, 7.6f), 
                                              make_float3(0.5f, 0.6f, 0.7f))));

    float2 f_data = make_float2(1.0f, 2.0f);
    float2 f_grad = make_float2(0.1f, 0.3f);

    FloatGrad<float2> f(f_data, f_grad);

    float2 g_data = make_float2(3.0f, 4.0f);
    float2 g_grad = make_float2(0.2f, 0.4f);

    FloatGrad<float2> g(g_data, g_grad);

    EXPECT_TRUE(float_eq(f * g_grad, FloatGrad<float2>(make_float2(0.2f, 0.8f), 
                                                       make_float2(0.02f, 0.12f))));

    EXPECT_TRUE(float_eq(f * g, FloatGrad<float2>(make_float2(3.0f, 8.0f), 
                                                  make_float2(0.5f, 2.0f))));

    EXPECT_TRUE(float_eq(g / f, FloatGrad<float2>(make_float2(3.0f, 2.0f), 
                                                  make_float2(-0.1f / 1, -0.4f / 4))));

}

TEST(FloatGradHelperMathTest, MinMaxFunctions) {
    float4 a_data = make_float4(1.0f, 2.0f, 3.0f, 4.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.3f, 0.4f);

    float4 b_data = make_float4(-2.0f, 3.0f, 2.5f, 5.0f);
    float4 b_grad = make_float4(-0.2f, -0.4f, -0.6f, -0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    FloatGrad<float4> c = fminf(a, b);

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(-2.0f, 2.0f, 2.5f, 4.0f), 
                                              make_float4(-0.2f, 0.2f, -0.6f, 0.4f))));

    float4 d = make_float4(1.0f, 1.0f, -3.0f, 5.0f);

    FloatGrad<float4> e = fminf(c, d);

    EXPECT_TRUE(float_eq(e, FloatGrad<float4>(make_float4(-2.0f, 1.0f, -3.0f, 4.0f), 
                                              make_float4(-0.2f, 0.0f, -0.0f, 0.4f))));

    FloatGrad<float4> f = fmaxf(a, b);

    EXPECT_TRUE(float_eq(f, FloatGrad<float4>(make_float4(1.0f, 3.0f, 3.0f, 5.0f), 
                                              make_float4(0.1f, -0.4f, 0.3f, -0.8f))));

    FloatGrad<float4> g = fmaxf(d, c);

    EXPECT_TRUE(float_eq(g, FloatGrad<float4>(make_float4(1.0f, 2.0f, 2.5f, 5.0f), 
                                              make_float4(0.0f, 0.2f, -0.6f, 0.0f))));

}

TEST(FloatGradHelperMathTest, LerpFunctions) {
    float4 a_data = make_float4(1.0f, 2.0f, 3.0f, 4.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.3f, 0.4f);

    float4 b_data = make_float4(2.0f, 4.0f, 6.0f, 8.0f);
    float4 b_grad = make_float4(0.2f, 0.4f, 0.6f, 0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    EXPECT_TRUE(float_eq(lerp(a.x, 3.0f, 0.5f), 
                         FloatGrad<float>(2.0f, 0.05f)));

    FloatGrad<float> t(0.1f, 1.0f);

    FloatGrad<float4> c = lerp(a, b, t);

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(1.1f, 2.2f, 3.3f, 4.4f), 
                                              make_float4(0.1f + 1.01f, 0.2f + 2.02f, 0.3f + 3.03f, 0.4f + 4.04f))));


}
