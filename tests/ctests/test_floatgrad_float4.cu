#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "hip/hip_vector_types.h"
#include "test_utils.h"

TEST(FloatGradFloat4, MakeFloat4) {
    FloatGrad<float> ax(1.0f, 0.1f);
    FloatGrad<float> ay(2.0f, 0.2f);
    FloatGrad<float> az(4.0f, 0.4f);
    FloatGrad<float> aw(6.0f, 0.6f);
    FloatGrad<float4> a = make_float4(ax, ay, az, aw);

    EXPECT_TRUE(float_eq(a, FloatGrad<float4>(make_float4(1.0f, 2.0f, 4.0f, 6.0f), 
                                              make_float4(0.1f, 0.2f, 0.4f, 0.6f))));

}

TEST(FloatGradFloat4, VectorElementAccess) {
    float4 a_data = make_float4(1.0f, 2.0f, 5.0f, -4.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.5f, -0.4f);
    float4 b_data = make_float4(3.0f, 4.0f, 6.0f, 8.0f);
    float4 b_grad = make_float4(0.3f, 0.4f, 0.6f, 0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    EXPECT_TRUE(float_eq(a.x, FloatGrad<float>(1.0f, 0.1f)));
    EXPECT_TRUE(float_eq(a.y, FloatGrad<float>(2.0f, 0.2f)));
    EXPECT_TRUE(float_eq(a.z, FloatGrad<float>(5.0f, 0.5f)));
    EXPECT_TRUE(float_eq(a.w, FloatGrad<float>(-4.0f, -0.4f)));

    a.w -= b.w;

    EXPECT_TRUE(float_eq(a, FloatGrad<float4>(float4{1.0f, 2.0f, 5.0f, -12.0f}, 
                                              float4{0.1f, 0.2f, 0.5f, -1.20f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float4>(float4{3.0f, 4.0f, 6.0f, 8.0f}, 
                                              float4{0.3f, 0.4f, 0.6f, 0.8f})));

    a.y *= b.y;

    EXPECT_TRUE(float_eq(a, FloatGrad<float4>(float4{1.0f, 8.0f, 5.0f, -12.0f}, 
                                              float4{0.1f, 1.6f, 0.5f, -1.20f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float4>(float4{3.0f, 4.0f, 6.0f, 8.0f}, 
                                              float4{0.3f, 0.4f, 0.6f, 0.8f})));
}

TEST(FloatGradFloat4, VectorOperators) {
    float4 a_data = make_float4(1.0f, 2.0f, 3.0f, 2.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.3f, 0.2f);
    float4 b_data = make_float4(3.0f, 4.0f, 6.0f, 8.0f);
    float4 b_grad = make_float4(0.3f, 0.4f, 0.6f, 0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    auto c = b - a;

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(2.0f, 2.0f, 3.0f, 6.0f), 
                                              make_float4(0.2f, 0.2f, 0.3f, 0.6f))));

}
