#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "hip/hip_vector_types.h"
#include "test_utils.h"

TEST(FloatGradFloat4, MakeFloat4) {
    FloatGrad<float> ax(1.0f, 0.1f);
    FloatGrad<float> ay(2.0f, 0.2f);
    FloatGrad<float> az(4.0f, 0.4f);
    FloatGrad<float> aw(6.0f, 0.6f);
    FloatGrad<float4> a = make_float4(ax, ay, az, aw);

    EXPECT_TRUE(float_eq(a, FloatGrad<float4>(make_float4(1.0f, 2.0f, 4.0f, 6.0f), 
                                              make_float4(0.1f, 0.2f, 0.4f, 0.6f))));

}

TEST(FloatGradFloat4, Float4Ctor) {
    FloatGrad<float> ax(1.0f, 0.1f);
    float ay = 2.0f;
    FloatGrad<float> az(4.0f, 0.4f);
    float aw = 6.0f;
    FloatGrad<float4> a(ax, ay, az, aw);

    EXPECT_TRUE(float_eq(a, FloatGrad<float4>(make_float4(1.0f, 2.0f, 4.0f, 6.0f), 
                                              make_float4(0.1f, 0.0f, 0.4f, 0.0f))));

}

TEST(FloatGradFloat4, VectorElementAccess) {
    float4 a_data = make_float4(1.0f, 2.0f, 5.0f, -4.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.5f, -0.4f);
    float4 b_data = make_float4(3.0f, 4.0f, 6.0f, 8.0f);
    float4 b_grad = make_float4(0.3f, 0.4f, 0.6f, 0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    EXPECT_TRUE(float_eq(a.x, FloatGrad<float>(1.0f, 0.1f)));
    EXPECT_TRUE(float_eq(a.y, FloatGrad<float>(2.0f, 0.2f)));
    EXPECT_TRUE(float_eq(a.z, FloatGrad<float>(5.0f, 0.5f)));
    EXPECT_TRUE(float_eq(a.w, FloatGrad<float>(-4.0f, -0.4f)));

    a.w -= b.w;

    EXPECT_TRUE(float_eq(a, FloatGrad<float4>(float4{1.0f, 2.0f, 5.0f, -12.0f}, 
                                              float4{0.1f, 0.2f, 0.5f, -1.20f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float4>(float4{3.0f, 4.0f, 6.0f, 8.0f}, 
                                              float4{0.3f, 0.4f, 0.6f, 0.8f})));

    a.y *= b.y;

    EXPECT_TRUE(float_eq(a, FloatGrad<float4>(float4{1.0f, 8.0f, 5.0f, -12.0f}, 
                                              float4{0.1f, 1.6f, 0.5f, -1.20f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float4>(float4{3.0f, 4.0f, 6.0f, 8.0f}, 
                                              float4{0.3f, 0.4f, 0.6f, 0.8f})));

    FloatGrad<const float4> c(a_data, a_grad);
    FloatGradRef<const float4> d(&b_data, &b_grad);

    EXPECT_TRUE(float_eq(c.x, FloatGrad<float>(1.0f, 0.1f)));
    EXPECT_TRUE(float_eq(c.y, FloatGrad<float>(2.0f, 0.2f)));
    EXPECT_TRUE(float_eq(c.z, FloatGrad<float>(5.0f, 0.5f)));
    EXPECT_TRUE(float_eq(c.w, FloatGrad<float>(-4.0f, -0.4f)));
    EXPECT_TRUE(float_eq(d.x, FloatGrad<float>(3.0f, 0.3f)));
    EXPECT_TRUE(float_eq(d.y, FloatGrad<float>(4.0f, 0.4f)));
    EXPECT_TRUE(float_eq(d.z, FloatGrad<float>(6.0f, 0.6f)));
    EXPECT_TRUE(float_eq(d.w, FloatGrad<float>(8.0f, 0.8f)));
}

TEST(FloatGradFloat4, VectorOperators) {
    float4 a_data = make_float4(1.0f, 2.0f, 3.0f, 2.0f);
    float4 a_grad = make_float4(0.1f, 0.2f, 0.3f, 0.2f);
    float4 b_data = make_float4(3.0f, 4.0f, 6.0f, 8.0f);
    float4 b_grad = make_float4(0.3f, 0.4f, 0.6f, 0.8f);

    FloatGrad<float4> a(a_data, a_grad);
    FloatGrad<float4> b(b_data, b_grad);

    auto c = b - a;

    EXPECT_TRUE(float_eq(c, FloatGrad<float4>(make_float4(2.0f, 2.0f, 3.0f, 6.0f), 
                                              make_float4(0.2f, 0.2f, 0.3f, 0.6f))));

}
