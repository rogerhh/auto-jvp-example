#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"

// Vector comparators only for testing purposes
__host__ __device__
bool operator==(const float2& a, const float2& b) {
    return a.x == b.x && a.y == b.y;
}

__host__ __device__
bool operator==(const float3& a, const float3& b) {
    return a.x == b.x && a.y == b.y && a.z == b.z;
}

__host__ __device__
bool operator==(const float4& a, const float4& b) {
    return a.x == b.x && a.y == b.y && a.z == b.z && a.w == b.w;
}


TEST(FloatGradTest, ScalarOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);
    
    FloatGrad<float> c = a + b;
    EXPECT_FLOAT_EQ(c.data, 7.0f);
    EXPECT_FLOAT_EQ(c.grad, 3.0f);

    FloatGrad<float> d = a - b;
    EXPECT_FLOAT_EQ(d.data, -1.0f);
    EXPECT_FLOAT_EQ(d.grad, -1.0f);

    FloatGrad<float> e = a * b;
    EXPECT_FLOAT_EQ(e.data, 12.0f);
    EXPECT_FLOAT_EQ(e.grad, 10.0f); // 3 * 4 + 1 * 2

    FloatGrad<float> f = a / b;
    EXPECT_FLOAT_EQ(f.data, 0.75f);
    EXPECT_FLOAT_EQ(f.grad, -0.125f); // (1 * 4 - 3 * 2) / (4 * 4)

    FloatGrad<float> g = sqrtf(b);
    EXPECT_FLOAT_EQ(g.data, 2.0f);
    EXPECT_FLOAT_EQ(g.grad, 0.5f); // 2 * 1/2 * 1/sqrt(4)

    // Comparators
    EXPECT_TRUE(a < b);
    EXPECT_FALSE(a > b);
    EXPECT_TRUE(a <= b);
    EXPECT_FALSE(a >= b);
    EXPECT_FALSE(a == b);
    EXPECT_TRUE(a != b);

    FloatGrad<float> m(5.0f, 10.0f);
    FloatGrad<float> n(5.0f, 7.0f);

    EXPECT_TRUE(m == n); // Same data, different grad
    EXPECT_FALSE(m != n); // Same data, different grad
    EXPECT_FALSE(m.eq(n)); // Should be false since grad is different

}

TEST(FloatGradTest, ScalarCompoundOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);

    a += b;
    EXPECT_FLOAT_EQ(a.data, 7.0f);
    EXPECT_FLOAT_EQ(a.grad, 3.0f);

    a *= b;
    EXPECT_FLOAT_EQ(a.data, 28.0f);
    EXPECT_FLOAT_EQ(a.grad, 26.0f); // 7 * 2 + 3 * 4

    a -= b;
    EXPECT_FLOAT_EQ(a.data, 24.0f);
    EXPECT_FLOAT_EQ(a.grad, 24.0f);

    a /= b;
    EXPECT_FLOAT_EQ(a.data, 6.0f);
    EXPECT_FLOAT_EQ(a.grad, 3.0f); // (24 * 4 - 24 * 2) / (4 * 4)
}

TEST(FloatGradTest, ScalarArrayOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);

    for(int i = 0; i < 10; i++) {
        FloatGrad<float> ai{a_data[i], a_grad[i]};
        FloatGrad<float> bi{b_data[i], b_grad[i]};

        EXPECT_TRUE((ai + bi).eq(a[i] + b[i]));
        EXPECT_TRUE((ai - bi).eq(a[i] - b[i]));
        EXPECT_TRUE((ai * bi).eq(a[i] * b[i]));
        EXPECT_TRUE((ai / bi).eq(a[i] / b[i]));
        EXPECT_TRUE(sqrtf(bi).eq(sqrtf<float>(b[i])));

        EXPECT_TRUE(!((ai < bi) ^ (a[i] < b[i])));
        EXPECT_TRUE(!((ai > bi) ^ (a[i] > b[i])));
        EXPECT_TRUE(!((ai <= bi) ^ (a[i] <= b[i])));
        EXPECT_TRUE(!((ai >= bi) ^ (a[i] >= b[i])));
    }

}

TEST(FloatGradTest, ScalarArrayCompoundOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};
    float c_data[10] = {-5.0f, -4.0f, -3.0f, -2.0f, -1.0f,
                            0.0f, 1.0f, 2.0f, 3.0f, 4.0f};
    float c_grad[10] = {-1.0f, -0.9f, -0.8f, -0.7f, -0.6f,
                            -0.5f, -0.4f, -0.3f, -0.2f, -0.1f};

    float c_copy_data[10];
    float c_copy_grad[10];

    for(int i = 0; i < 10; i++) {
        c_copy_data[i] = c_data[i];
        c_copy_grad[i] = c_grad[i];
    }

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);
    FloatGradArray<float> c(c_data, c_grad);

    for(int i = 0; i < 10; i++) {
        c[i] += a[i] * b[i];

        EXPECT_TRUE(c[i].eq((FloatGrad<float>{c_copy_data[i], c_copy_grad[i]}
                + FloatGrad<float>{a_data[i], a_grad[i]} 
                * FloatGrad<float>{b_data[i], b_grad[i]})));
    }

}

TEST(FloatGradTest, VectorElementAccess) {
    float2 a_data = make_float2(1.0f, 2.0f);
    float2 a_grad = make_float2(0.1f, 0.2f);
    float2 b_data = make_float2(3.0f, 4.0f);
    float2 b_grad = make_float2(0.3f, 0.4f);

    FloatGrad<float2> a(a_data, a_grad);
    FloatGrad<float2> b(b_data, b_grad);

    a.x += b.x;

    EXPECT_TRUE(a.eq(FloatGrad<float2>(float2{4.0f, 2.0f}, float2{0.4f, 0.2f})));
    EXPECT_TRUE(b.eq(FloatGrad<float2>(float2{3.0f, 4.0f}, float2{0.3f, 0.4f})));

    a.y *= b.y;

    EXPECT_TRUE(a.eq(FloatGrad<float2>(float2{4.0f, 8.0f}, float2{0.4f, 1.6f})));
    EXPECT_TRUE(b.eq(FloatGrad<float2>(float2{3.0f, 4.0f}, float2{0.3f, 0.4f})));
}

TEST(FloatGradTest, VectorOperators) {

}

