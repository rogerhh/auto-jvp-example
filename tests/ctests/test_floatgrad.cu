#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "test_utils.h"

TEST(FloatGradTest, RefConstructorAndAssignment) {
    float a_data = 3.0f, a_grad = 1.0f;
    const float b_data = 4.0f;
    const float b_grad = 2.0f;

    FloatGradRef<float> a(&a_data, &a_grad);
    FloatGradRef<const float> b(&b_data, &b_grad);
    FloatGradRef<const float> c(a);
    FloatGradRef<const float> d(b);
    // FloatGradRef<float> e(b);   // This should not compile

    EXPECT_TRUE(float_eq(a, FloatGrad<float>(3.0f, 1.0f)));
    EXPECT_TRUE(float_eq(b, FloatGrad<float>(4.0f, 2.0f)));
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(3.0f, 1.0f)));
    EXPECT_TRUE(float_eq(d, FloatGrad<float>(4.0f, 2.0f)));

    // Ref to Ref
    float f_data = 5.0f, f_grad = 3.0f;
    FloatGradRef<float> f(&f_data, &f_grad);
    a = f;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(5.0f, 3.0f)));

    // Const Ref to Ref
    a = d;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(4.0f, 2.0f)));

    // Val to Ref
    float g_data = 6.0f, g_grad = 4.0f;
    FloatGrad<float> g(g_data, g_grad);
    a = g;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(6.0f, 4.0f)));

    // Const Val to Ref
    const float h_data = 7.0f;
    const float h_grad = 5.0f;
    FloatGrad<const float> h(h_data, h_grad);
    a = h;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(7.0f, 5.0f)));

}

TEST(FloatGradTest, ValConstructorAndAssignment) {
    float a_data = 3.0f, a_grad = 1.0f;
    const float b_data = 4.0f;
    const float b_grad = 2.0f;

    FloatGrad<float> a(a_data, a_grad);
    FloatGrad<const float> b(b_data, b_grad);
    FloatGrad<const float> c(a);
    FloatGrad<const float> d(b);
    FloatGrad<float> e(b);   // This should not compile

    EXPECT_TRUE(float_eq(a, FloatGrad<float>(3.0f, 1.0f)));
    EXPECT_TRUE(float_eq(b, FloatGrad<float>(4.0f, 2.0f)));
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(3.0f, 1.0f)));
    EXPECT_TRUE(float_eq(d, FloatGrad<float>(4.0f, 2.0f)));
    EXPECT_TRUE(float_eq(e, FloatGrad<float>(4.0f, 2.0f)));

    // Val to Val
    float f_data = 5.0f, f_grad = 3.0f;
    FloatGrad<float> f(f_data, f_grad);
    a = f;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(5.0f, 3.0f)));

    // Const Val to Val
    a = d;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(4.0f, 2.0f)));

    // Ref to Val
    float g_data = 6.0f, g_grad = 4.0f;
    FloatGradRef<float> g(&g_data, &g_grad);
    a = g;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(6.0f, 4.0f)));

    // Ref to Const Val
    FloatGradRef<const float> h(&a_data, &a_grad);
    a = h;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(3.0f, 1.0f)));
}

TEST(FloatGradTest, ScalarOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);
     
    FloatGrad<float> c = a + b;
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(7.0f, 3.0f)));

    FloatGrad<float> d = a - b;
    EXPECT_TRUE(float_eq(d, FloatGrad<float>(-1.0f, -1.0f)));

    FloatGrad<float> e = a * b;
    // grad = 3 * 4 + 1 * 2
    EXPECT_TRUE(float_eq(e, FloatGrad<float>(12.0f, 10.0f))); 

    FloatGrad<float> f = a / b;
    // grad = (3 * 4 - 1 * 2) / (4 * 4)
    EXPECT_TRUE(float_eq(f, FloatGrad<float>(0.75f, -0.125f))) << "Expected: 0.75, -0.125, Got: " << f.data() << ", " << f.grad(); 

    FloatGrad<float> g = sqrtf(b);
    // sqrt(4) = 2, grad = 1/2 * 1/sqrt(4)
    EXPECT_TRUE(float_eq(g, FloatGrad<float>(2.0f, 0.5f))); 

    // Comparators
    EXPECT_TRUE(a < b);
    EXPECT_FALSE(a > b);
    EXPECT_TRUE(a <= b);
    EXPECT_FALSE(a >= b);
    EXPECT_FALSE(a == b);
    EXPECT_TRUE(a != b);

    FloatGrad<float> m(5.0f, 10.0f);
    FloatGrad<float> n(5.0f, 7.0f);

    EXPECT_TRUE(m == n); // Same data, different grad
    EXPECT_FALSE(m != n); // Same data, different grad
    EXPECT_FALSE(float_eq(m, n)); // Should be false since grad is different

}

TEST(FloatGradTest, ScalarCompoundOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);

    a += b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(7.0f, 3.0f)));

    a *= b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(28.0f, 26.0f))); // 7 * 4 + 3 * 2

    a -= b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(24.0f, 24.0f)));

    a /= b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(6.0f, 3.0f))); // (24 * 4 - 24 * 2) / (4 * 4)
}

