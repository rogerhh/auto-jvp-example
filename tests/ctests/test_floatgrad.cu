#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "test_utils.h"


TEST(FloatGradTest, ScalarOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);
     
    FloatGrad<float> c = a + b;
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(7.0f, 3.0f)));

    FloatGrad<float> d = a - b;
    EXPECT_TRUE(float_eq(d, FloatGrad<float>(-1.0f, -1.0f)));

    FloatGrad<float> e = a * b;
    // grad = 3 * 4 + 1 * 2
    EXPECT_TRUE(float_eq(e, FloatGrad<float>(12.0f, 10.0f))); 

    FloatGrad<float> f = a / b;
    // grad = (3 * 4 - 1 * 2) / (4 * 4)
    EXPECT_TRUE(float_eq(f, FloatGrad<float>(0.75f, -0.125f))) << "Expected: 0.75, -0.125, Got: " << f.data() << ", " << f.grad(); 

    FloatGrad<float> g = sqrtf(b);
    // sqrt(4) = 2, grad = 1/2 * 1/sqrt(4)
    EXPECT_TRUE(float_eq(g, FloatGrad<float>(2.0f, 0.5f))); 

    // Comparators
    EXPECT_TRUE(a < b);
    EXPECT_FALSE(a > b);
    EXPECT_TRUE(a <= b);
    EXPECT_FALSE(a >= b);
    EXPECT_FALSE(a == b);
    EXPECT_TRUE(a != b);

    FloatGrad<float> m(5.0f, 10.0f);
    FloatGrad<float> n(5.0f, 7.0f);

    EXPECT_TRUE(m == n); // Same data, different grad
    EXPECT_FALSE(m != n); // Same data, different grad
    EXPECT_FALSE(float_eq(m, n)); // Should be false since grad is different

}

TEST(FloatGradTest, ScalarCompoundOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);

    a += b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(7.0f, 3.0f)));

    a *= b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(28.0f, 26.0f))); // 7 * 4 + 3 * 2

    a -= b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(24.0f, 24.0f)));

    a /= b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(6.0f, 3.0f))); // (24 * 4 - 24 * 2) / (4 * 4)
}

TEST(FloatGradTest, ScalarArrayOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);

    for(int i = 0; i < 10; i++) {
        FloatGrad<float> ai{a_data[i], a_grad[i]};
        FloatGrad<float> bi{b_data[i], b_grad[i]};

        EXPECT_TRUE(float_eq(ai, a[i]));
        EXPECT_TRUE(float_eq(bi, b[i]));
        EXPECT_TRUE(float_eq(ai + bi, a[i] + b[i]));
        EXPECT_TRUE(float_eq(ai - bi, a[i] - b[i]));
        EXPECT_TRUE(float_eq(ai * bi, a[i] * b[i]));
        EXPECT_TRUE(float_eq(ai / bi, a[i] / b[i]));
        EXPECT_TRUE(float_eq(sqrtf(bi), sqrtf(b[i])));

        EXPECT_TRUE(!((ai < bi) ^ (a[i] < b[i])));
        EXPECT_TRUE(!((ai > bi) ^ (a[i] > b[i])));
        EXPECT_TRUE(!((ai <= bi) ^ (a[i] <= b[i])));
        EXPECT_TRUE(!((ai >= bi) ^ (a[i] >= b[i])));
    }

}

TEST(FloatGradTest, ScalarArrayCompoundOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};
    float c_data[10] = {-5.0f, -4.0f, -3.0f, -2.0f, -1.0f,
                            0.0f, 1.0f, 2.0f, 3.0f, 4.0f};
    float c_grad[10] = {-1.0f, -0.9f, -0.8f, -0.7f, -0.6f,
                            -0.5f, -0.4f, -0.3f, -0.2f, -0.1f};

    float c_copy_data[10];
    float c_copy_grad[10];

    for(int i = 0; i < 10; i++) {
        c_copy_data[i] = c_data[i];
        c_copy_grad[i] = c_grad[i];
    }

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);
    FloatGradArray<float> c(c_data, c_grad);

    for(int i = 0; i < 10; i++) {
        c[i] += a[i] * b[i];

        EXPECT_TRUE(float_eq(c[i], 
                    FloatGrad<float>(c_copy_data[i] + a_data[i] * b_data[i],
                    c_copy_grad[i] + a_grad[i] * b_data[i] + a_data[i] * b_grad[i])));
    }

}

