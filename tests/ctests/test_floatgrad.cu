#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "float_grad_float2.h"

__host__ __device__
bool float_eq(float a, float b, float eps = 1e-6f) {
    return fabs(a - b) < eps;
}

__host__ __device__
bool float_eq(float2 a, float2 b, float eps = 1e-6f) {
    return float_eq(a.x, b.x, eps) && float_eq(a.y, b.y, eps);
}

__host__ __device__
bool float_eq(float3 a, float3 b, float eps = 1e-6f) {
    return float_eq(a.x, b.x, eps) && float_eq(a.y, b.y, eps) && float_eq(a.z, b.z, eps);
}

__host__ __device__
bool float_eq(float4 a, float4 b, float eps = 1e-6f) {
    return float_eq(a.x, b.x, eps) && float_eq(a.y, b.y, eps) &&
           float_eq(a.z, b.z, eps) && float_eq(a.w, b.w, eps);
}

template <typename T1, typename T2,
          typename = std::enable_if_t<is_float_grad<T1>::value
                                      && is_float_grad<T2>::value>>
__host__ __device__
bool float_eq(const T1& a, const T2& b, float eps = 1e-6f) {
    return float_eq(a.data(), b.data(), eps) && float_eq(a.grad(), b.grad(), eps);
}

TEST(FloatGradTest, ScalarOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);
     
    FloatGrad<float> c = a + b;
    EXPECT_TRUE(float_eq(c, FloatGrad<float>(7.0f, 3.0f)));

    FloatGrad<float> d = a - b;
    EXPECT_TRUE(float_eq(d, FloatGrad<float>(-1.0f, -1.0f)));

    FloatGrad<float> e = a * b;
    // grad = 3 * 4 + 1 * 2
    EXPECT_TRUE(float_eq(e, FloatGrad<float>(12.0f, 10.0f))); 

    FloatGrad<float> f = a / b;
    // grad = (3 * 4 - 1 * 2) / (4 * 4)
    EXPECT_TRUE(float_eq(f, FloatGrad<float>(0.75f, -0.125f))) << "Expected: 0.75, -0.125, Got: " << f.data() << ", " << f.grad(); 

    FloatGrad<float> g = sqrtf(b);
    // sqrt(4) = 2, grad = 1/2 * 1/sqrt(4)
    EXPECT_TRUE(float_eq(g, FloatGrad<float>(2.0f, 0.5f))); 

    // Comparators
    EXPECT_TRUE(a < b);
    EXPECT_FALSE(a > b);
    EXPECT_TRUE(a <= b);
    EXPECT_FALSE(a >= b);
    EXPECT_FALSE(a == b);
    EXPECT_TRUE(a != b);

    FloatGrad<float> m(5.0f, 10.0f);
    FloatGrad<float> n(5.0f, 7.0f);

    EXPECT_TRUE(m == n); // Same data, different grad
    EXPECT_FALSE(m != n); // Same data, different grad
    EXPECT_FALSE(float_eq(m, n)); // Should be false since grad is different

}

TEST(FloatGradTest, ScalarCompoundOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);

    a += b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(7.0f, 3.0f)));

    a *= b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(28.0f, 26.0f))); // 7 * 4 + 3 * 2

    a -= b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(24.0f, 24.0f)));

    a /= b;
    EXPECT_TRUE(float_eq(a, FloatGrad<float>(6.0f, 3.0f))); // (24 * 4 - 24 * 2) / (4 * 4)
}

TEST(FloatGradTest, ScalarArrayOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);

    for(int i = 0; i < 10; i++) {
        FloatGrad<float> ai{a_data[i], a_grad[i]};
        FloatGrad<float> bi{b_data[i], b_grad[i]};

        EXPECT_TRUE(float_eq(ai, a[i]));
        EXPECT_TRUE(float_eq(bi, b[i]));
        EXPECT_TRUE(float_eq(ai + bi, a[i] + b[i]));
        EXPECT_TRUE(float_eq(ai - bi, a[i] - b[i]));
        EXPECT_TRUE(float_eq(ai * bi, a[i] * b[i]));
        EXPECT_TRUE(float_eq(ai / bi, a[i] / b[i]));
        EXPECT_TRUE(float_eq(sqrtf(bi), sqrtf(b[i])));

        EXPECT_TRUE(!((ai < bi) ^ (a[i] < b[i])));
        EXPECT_TRUE(!((ai > bi) ^ (a[i] > b[i])));
        EXPECT_TRUE(!((ai <= bi) ^ (a[i] <= b[i])));
        EXPECT_TRUE(!((ai >= bi) ^ (a[i] >= b[i])));
    }

}

TEST(FloatGradTest, ScalarArrayCompoundOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};
    float c_data[10] = {-5.0f, -4.0f, -3.0f, -2.0f, -1.0f,
                            0.0f, 1.0f, 2.0f, 3.0f, 4.0f};
    float c_grad[10] = {-1.0f, -0.9f, -0.8f, -0.7f, -0.6f,
                            -0.5f, -0.4f, -0.3f, -0.2f, -0.1f};

    float c_copy_data[10];
    float c_copy_grad[10];

    for(int i = 0; i < 10; i++) {
        c_copy_data[i] = c_data[i];
        c_copy_grad[i] = c_grad[i];
    }

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);
    FloatGradArray<float> c(c_data, c_grad);

    for(int i = 0; i < 10; i++) {
        c[i] += a[i] * b[i];

        EXPECT_TRUE(c[i].eq((FloatGrad<float>{c_copy_data[i], c_copy_grad[i]}
                + FloatGrad<float>{a_data[i], a_grad[i]} 
                * FloatGrad<float>{b_data[i], b_grad[i]})));
    }

}

TEST(FloatGradTest, VectorElementAccess) {
    float2 a_data = make_float2(1.0f, 2.0f);
    float2 a_grad = make_float2(0.1f, 0.2f);
    float2 b_data = make_float2(3.0f, 4.0f);
    float2 b_grad = make_float2(0.3f, 0.4f);

    FloatGrad<float2> a(a_data, a_grad);
    FloatGrad<float2> b(b_data, b_grad);

    EXPECT_TRUE(float_eq(a.x, FloatGrad<float>(1.0f, 0.1f)));
    EXPECT_TRUE(float_eq(a.y, FloatGrad<float>(2.0f, 0.2f)));

    a.x += b.x;

    EXPECT_TRUE(float_eq(a, FloatGrad<float2>(float2{4.0f, 2.0f}, float2{0.4f, 0.2f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(float2{3.0f, 4.0f}, float2{0.3f, 0.4f})));

    a.y *= b.y;

    EXPECT_TRUE(float_eq(a, FloatGrad<float2>(float2{4.0f, 8.0f}, float2{0.4f, 1.6f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float2>(float2{3.0f, 4.0f}, float2{0.3f, 0.4f})));
}

