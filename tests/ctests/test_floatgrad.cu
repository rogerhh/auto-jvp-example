#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"


TEST(FloatGradTest, ScalarOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);
    
    FloatGrad<float> c = a + b;
    EXPECT_FLOAT_EQ(c.data, 7.0f);
    EXPECT_FLOAT_EQ(c.grad, 3.0f);

    FloatGrad<float> d = a - b;
    EXPECT_FLOAT_EQ(d.data, -1.0f);
    EXPECT_FLOAT_EQ(d.grad, -1.0f);

    FloatGrad<float> e = a * b;
    EXPECT_FLOAT_EQ(e.data, 12.0f);
    EXPECT_FLOAT_EQ(e.grad, 10.0f); // 3 * 4 + 1 * 2

    FloatGrad<float> f = a / b;
    EXPECT_FLOAT_EQ(f.data, 0.75f);
    EXPECT_FLOAT_EQ(f.grad, -0.125f); // (1 * 4 - 3 * 2) / (4 * 4)

    FloatGrad<float> g = sqrtf(b);
    EXPECT_FLOAT_EQ(g.data, 2.0f);
    EXPECT_FLOAT_EQ(g.grad, 0.5f); // 2 * 1/2 * 1/sqrt(4)

    // Comparators
    EXPECT_TRUE(a < b);
    EXPECT_FALSE(a > b);
    EXPECT_TRUE(a <= b);
    EXPECT_FALSE(a >= b);
    EXPECT_FALSE(a == b);
    EXPECT_TRUE(a != b);

    FloatGrad<float> m(5.0f, 10.0f);
    FloatGrad<float> n(5.0f, 7.0f);

    EXPECT_TRUE(m == n); // Same data, different grad
    EXPECT_FALSE(m != n); // Same data, different grad

}

TEST(FloatGradTest, ScalarCompoundOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);

    a += b;
    EXPECT_FLOAT_EQ(a.data, 7.0f);
    EXPECT_FLOAT_EQ(a.grad, 3.0f);

    a *= b;
    EXPECT_FLOAT_EQ(a.data, 28.0f);
    EXPECT_FLOAT_EQ(a.grad, 26.0f); // 7 * 2 + 3 * 4

    a -= b;
    EXPECT_FLOAT_EQ(a.data, 24.0f);
    EXPECT_FLOAT_EQ(a.grad, 24.0f);

    a /= b;
    EXPECT_FLOAT_EQ(a.data, 6.0f);
    EXPECT_FLOAT_EQ(a.grad, 3.0f); // (24 * 4 - 24 * 2) / (4 * 4)
}

TEST(FloatGradTest, ScalarArrayOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);

    for(int i = 0; i < 10; i++) {
        FloatGrad<float> ai{a_data[i], a_grad[i]};
        FloatGrad<float> bi{b_data[i], b_grad[i]};

        EXPECT_TRUE(ai + bi == a[i] + b[i]);
        EXPECT_TRUE(ai - bi == a[i] - b[i]);
        EXPECT_TRUE(ai * bi == a[i] * b[i]);
        EXPECT_TRUE(ai / bi == a[i] / b[i]);
        EXPECT_TRUE(sqrtf(bi) == sqrtf<float>(b[i]));

        EXPECT_TRUE(!((ai < bi) ^ (a[i] < b[i])));
        EXPECT_TRUE(!((ai > bi) ^ (a[i] > b[i])));
        EXPECT_TRUE(!((ai <= bi) ^ (a[i] <= b[i])));
        EXPECT_TRUE(!((ai >= bi) ^ (a[i] >= b[i])));
    }

}

TEST(FloatGradTest, ScalarArrayCompoundOperators) {
    float a_data[10] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                            6.0f, 7.0f, 8.0f, 9.0f, 10.0f};
    float a_grad[10] = {1.0f, 0.9f, 0.8f, 0.7f, 0.6f,
                            0.5f, 0.4f, 0.3f, 0.2f, 0.1f};
    float b_data[10] = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f,
                            12.0f, 14.0f, 16.0f, 18.0f, 20.0f};
    float b_grad[10] = {0.5f, 0.4f, 0.3f, 0.2f, 0.1f,
                            0.05f, 0.04f, 0.03f, 0.02f, 0.01f};
    float c_data[10] = {-5.0f, -4.0f, -3.0f, -2.0f, -1.0f,
                            0.0f, 1.0f, 2.0f, 3.0f, 4.0f};
    float c_grad[10] = {-1.0f, -0.9f, -0.8f, -0.7f, -0.6f,
                            -0.5f, -0.4f, -0.3f, -0.2f, -0.1f};

    float c_copy_data[10];
    float c_copy_grad[10];

    for(int i = 0; i < 10; i++) {
        c_copy_data[i] = c_data[i];
        c_copy_grad[i] = c_grad[i];
    }

    FloatGradArray<float> a(a_data, a_grad);
    FloatGradArray<float> b(b_data, b_grad);
    FloatGradArray<float> c(c_data, c_grad);

    for(int i = 0; i < 10; i++) {
        c[i] += a[i] * b[i];

        EXPECT_TRUE(c[i] == (FloatGrad<float>{c_copy_data[i], c_copy_grad[i]}
                + FloatGrad<float>{a_data[i], a_grad[i]} 
                * FloatGrad<float>{b_data[i], b_grad[i]}));
    }

}

TEST(FloatGradTest, VectorOperators) {
    FloatGrad<float> a(3.0f, 1.0f);
    FloatGrad<float> b(4.0f, 2.0f);
    
    FloatGrad<float> c = a + b;
    EXPECT_FLOAT_EQ(c.data, 7.0f);
    EXPECT_FLOAT_EQ(c.grad, 3.0f);

    FloatGrad<float> d = a - b;
    EXPECT_FLOAT_EQ(d.data, -1.0f);
    EXPECT_FLOAT_EQ(d.grad, -1.0f);

    FloatGrad<float> e = a * b;
    EXPECT_FLOAT_EQ(e.data, 12.0f);
    EXPECT_FLOAT_EQ(e.grad, 10.0f); // 3 * 4 + 1 * 2

    FloatGrad<float> f = a / b;
    EXPECT_FLOAT_EQ(f.data, 0.75f);
    EXPECT_FLOAT_EQ(f.grad, -0.125f); // (1 * 4 - 3 * 2) / (4 * 4)

    FloatGrad<float> g = sqrtf(b);
    EXPECT_FLOAT_EQ(g.data, 2.0f);
    EXPECT_FLOAT_EQ(g.grad, 0.5f); // 2 * 1/2 * 1/sqrt(4)

    // Comparators
    EXPECT_TRUE(a < b);
    EXPECT_FALSE(a > b);
    EXPECT_TRUE(a <= b);
    EXPECT_FALSE(a >= b);
    EXPECT_FALSE(a == b);
    EXPECT_TRUE(a != b);

    FloatGrad<float> m(5.0f, 10.0f);
    FloatGrad<float> n(5.0f, 7.0f);

    EXPECT_TRUE(m == n); // Same data, different grad
    EXPECT_FALSE(m != n); // Same data, different grad

}
