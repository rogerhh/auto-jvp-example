#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>

#include "float_grad.h"
#include "hip/hip_vector_types.h"
#include "test_utils.h"

TEST(FloatGradFloat3, MakeFloat3) {
    FloatGrad<float> ax(1.0f, 0.1f);
    FloatGrad<float> ay(2.0f, 0.2f);
    FloatGrad<float> az(4.0f, 0.4f);
    FloatGrad<float3> a = make_float3(ax, ay, az);

    EXPECT_TRUE(float_eq(a, FloatGrad<float3>(make_float3(1.0f, 2.0f, 4.0f), 
                                              make_float3(0.1f, 0.2f, 0.4f))));

}

TEST(FloatGradFloat3, VectorElementAccess) {
    float3 a_data = make_float3(1.0f, 2.0f, 5.0f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.5f);
    float3 b_data = make_float3(3.0f, 4.0f, 6.0f);
    float3 b_grad = make_float3(0.3f, 0.4f, 0.6f);

    FloatGrad<float3> a(a_data, a_grad);
    FloatGrad<float3> b(b_data, b_grad);

    EXPECT_TRUE(float_eq(a.x, FloatGrad<float>(1.0f, 0.1f)));
    EXPECT_TRUE(float_eq(a.y, FloatGrad<float>(2.0f, 0.2f)));
    EXPECT_TRUE(float_eq(a.z, FloatGrad<float>(5.0f, 0.5f)));

    a.z += b.z;

    EXPECT_TRUE(float_eq(a, FloatGrad<float3>(float3{1.0f, 2.0f, 11.0f}, 
                                              float3{0.1f, 0.2f, 1.10f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float3>(float3{3.0f, 4.0f, 6.0f}, 
                                              float3{0.3f, 0.4f, 0.6f})));

    a.y *= b.y;

    EXPECT_TRUE(float_eq(a, FloatGrad<float3>(float3{1.0f, 8.0f, 11.0f}, 
                                              float3{0.1f, 1.6f, 1.10f})));
    EXPECT_TRUE(float_eq(b, FloatGrad<float3>(float3{3.0f, 4.0f, 6.0f}, 
                                              float3{0.3f, 0.4f, 0.6f})));
}

TEST(FloatGradFloat3, VectorOperators) {
    float3 a_data = make_float3(1.0f, 2.0f, 3.0f);
    float3 a_grad = make_float3(0.1f, 0.2f, 0.3f);
    float3 b_data = make_float3(3.0f, 4.0f, 6.0f);
    float3 b_grad = make_float3(0.3f, 0.4f, 0.6f);

    FloatGrad<float3> a(a_data, a_grad);
    FloatGrad<float3> b(b_data, b_grad);

    auto c = b / a;

    EXPECT_TRUE(float_eq(c, FloatGrad<float3>(make_float3(3.0f, 2.0f, 2.0f), 
                                              make_float3(0.0f, 0.0f, 0.0f))));

}
